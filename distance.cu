#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <cstdlib>
#include <omp.h>

//#define N (2 << 20)
//#define N (1<<10)
#define L 6
#define NT 1024

// CUDA Kernel
//   (x, y, z, d) are thrust vectors of size N
//   (p) is a device pointer array size 3
__global__ void distance(float *x, float *y, float *z, float *d, int i){
    int tx = threadIdx.x;
    int j  = threadIdx.x + blockDim.x * blockIdx.x;
    
    __shared__ float s_x[NT];
    __shared__ float s_y[NT];
    __shared__ float s_z[NT];

    s_x[tx] = x[j];
    s_y[tx] = y[j];
    s_z[tx] = z[j];
    __syncthreads();
    
    //so that the particle itself will not be selected as a nearest neighbor
    d[i] = 10000000.0;

    if (j != i)
    {
        d[j] = sqrt( (s_x[tx] - x[i]) * (s_x[tx] - x[i]) +
                     (s_y[tx] - y[i]) * (s_y[tx] - y[i]) +
                     (s_z[tx] - z[i]) * (s_z[tx] - z[i]));
    }
}

void obtain_densities(float *mx, float *my, float *mz, float *mradius, float *mdensity, int istart, int icount, int ikid, int N, float mass)
{

    int i, j;
    double ini, end;
    float radius,density;

    hipSetDevice(ikid);

    // Host vectors (x, y, z, r)
    thrust::host_vector<float> h_x(N), h_y(N), h_z(N), h_r(N);
 
    // Fillong host vectors (x, y, z) with random numbers
    /*    thrust::generate(h_x.begin(), h_x.end(), rand);
    thrust::generate(h_y.begin(), h_y.end(), rand);
    thrust::generate(h_z.begin(), h_z.end(), rand);*/
    std::fill(h_r.begin(), h_r.end(), 0);

    //copy into host vectors;
    for (i = 0; i < N; i++)
      {
	h_x[i] = mx[i];
	h_y[i] = my[i];
	h_z[i] = mz[i];
	
      }

    // Device vectors (x, y, z, r ) using the memory of the Host vectors
    thrust::device_vector<float> d_x = h_x, d_y = h_y, d_z = h_z, d_r = h_r; 
   
    // Raw pointer of the vectors to use a normal CUDA kernel (x, y, z, r)
    float *x = thrust::raw_pointer_cast( &d_x[0] );
    float *y = thrust::raw_pointer_cast( &d_y[0] );
    float *z = thrust::raw_pointer_cast( &d_z[0] );
    float *r = thrust::raw_pointer_cast( &d_r[0] );

    // CUDA kernel configuration
    int nthread = NT;
    int nblocks = ceil(N/nthread);

    // GPU procedure

    ini = omp_get_wtime();
    
    for(i = istart; i < istart+icount; i++)
    {
        distance <<< nblocks, nthread >>> (x,y,z,r,i);
        hipDeviceSynchronize();

        // Sorting d_r
        thrust::sort(d_r.begin(), d_r.end());
        
        // Copy data after sorting
        //  from d_r to h_r
        thrust::copy(d_r.begin(), d_r.begin() + L, h_r.begin());
	/*        for (j = 0; j < L; j++) {
            std::cout << h_r[j] << " ";
        }
        std::cout << std::endl;
	*/
	//need to read the L-1 element of the distances
	radius = d_r[L-1];
	density = (L-1)*mass*3.0/4.0/(radius*radius*radius);
	mradius[i] = radius;
	mdensity[i] = density;
    }

    end = omp_get_wtime();
    std::cout << "GPU time  per particle: " << (end-ini)/icount << std::endl;
    std::cout << "GPU total time: " << (end-ini) << std::endl;

    // CPU procedure
    /*
    std::vector<float> e(N);

    ini = omp_get_wtime();
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
        {
            e[j] = sqrt( (h_x[j] - h_x[i]) * (h_x[j] - h_x[i]) +
                         (h_y[j] - h_y[i]) * (h_y[j] - h_y[i]) +
                         (h_z[j] - h_z[i]) * (h_z[j] - h_z[i]));

        }
        std::sort(e.begin(), e.end());
        for (j = 0; j < L; j++) {
            std::cout << e[j] << " ";
        }
        std::cout << std::endl;
    }

    end = omp_get_wtime();
    std::cout << "CPU time  per particle: " << (end-ini)/N << std::endl;
    std::cout << "CPU total time: " << (end-ini) << std::endl;
    */

}
